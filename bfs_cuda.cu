
#include "hip/hip_runtime.h"


// #include <device_functions.h>
#include <hip/hip_runtime_api.h>

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <iostream>

using namespace std;
// #include <bits/stdc++.h>
// #include <conio.h>
#define NUM_NODES 24
// #define NUM_PERMUTATIONS 

typedef struct
{
	int start;     // Index of first adjacent node in Ea	
	int length;    // Number of adjacent nodes 
} Node;

typedef struct
{
	int id;
	string s;
	/* data */
} HashMap;

__global__ void CUDA_BFS_KERNEL(Node *Va, int *Ea, bool *Fa, bool *Xa, int *Ca,bool *done)
{

	int id = threadIdx.x + blockIdx.x * blockDim.x;
	if (id > NUM_NODES)
		return;


	if (Fa[id] == true && Xa[id] == false)
	{
		printf("%d ", id); //This printf gives the order of vertices in BFS	
		Fa[id] = false;
		Xa[id] = true;
		__syncthreads(); 
		int k = 0;
		int i;
		int start = Va[id].start;
		int end = start + Va[id].length;
		for (int i = start; i < end; i++) 
		{
			int nid = Ea[i];

			if (Xa[nid] == false)
			{
				Ca[nid] = Ca[id] + 1;
				Fa[nid] = true;
				*done = false;
			}

		}

	}

}

int count=0;
int edgeNumber = 0;
HashMap map[1000];
Node node[NUM_NODES];
int edges[NUM_NODES*(NUM_NODES-1)];

void swap(string a, int l, int i) 
{ 
    char temp; 
    temp = a[l];
    a[l] = a[i]; 
    a[i] = temp; 
} 


void permute(string a, int l, int r) 
{ 
   int i; 
   if (l == r) 
   {
   	 // cout<<a<<endl;
     map[count].id = count;
     map[count].s = a;
     count+=1;
   }
   else
   { 
       for (i = l; i <= r; i++) 
       { 
          char temp; 
		    temp = a[l];
		    a[l] = a[i]; 
		    a[i] = temp;
          permute(a, l+1, r); 
 
		    temp = a[l];
		    a[l] = a[i]; 
		    a[i] = temp; //backtrack 
       } 
   } 
}

void getneighbour(string s, int i)
{
	int mat[2][2];
	for(int j=0;j<2;j++)
	{
		for(int k=0;k<2;k++)
		{
			mat[j][k] = s[j*2+k]-'0';
		}
	}

	int posx,posy;
	for(int j=0;j<2;j++)
	{
		for(int k=0;k<2;k++)
		{
			if(mat[j][k] == 0)
			{
				posx = j;
				posy = k;
				break;
			}
		}
	}

	 if (posx == 0 && posy == 0) 
	 {
        int Temp[2][2];
        for(int j=0;j<2;j++)
        {
        	for(int k=0;k<2;k++)
        	{
        		Temp[j][k] = mat[j][k];
        	}
        }

        node[i].start = edgeNumber;
        node[i].length = 2;

        // Moving 0 to the right
        Temp[posx][posy] = Temp[posx][posy+1];
        Temp[posx][posy+1] = 0;

        string s1 = "abcd";

        for(int j=0;j<2;j++)
        {
        	for(int k=0;k<2;k++)
        	{
        		s1[j*2+k] = Temp[j][k]+'0';
        	}
        }

        for(int j=0;j<24;j++)
        {
        	if (map[j].s == s1)
        	{
        		edges[edgeNumber++] = map[j].id;
        	}
        }


        // int Temp[2][2];
        for(int j=0;j<2;j++)
        {
        	for(int k=0;k<2;k++)
        	{
        		Temp[j][k] = mat[j][k];
        	}
        }

        // Moving 0 to the bottom
        Temp[posx][posy] = Temp[posx+1][posy];
        Temp[posx+1][posy] = 0;

        // s1 = "abcd";

        for(int j=0;j<2;j++)
        {
        	for(int k=0;k<2;k++)
        	{
        		s1[j*2+k] = Temp[j][k]+'0';
        	}
        }

        for(int j=0;j<24;j++)
        {
        	if (map[j].s == s1)
        	{
        		edges[edgeNumber++] = map[j].id;
        	}
        }
    }


    if (posx == 0 && posy == 1) 
	{
        int Temp[2][2];
        for(int j=0;j<2;j++)
        {
        	for(int k=0;k<2;k++)
        	{
        		Temp[j][k] = mat[j][k];
        	}
        }

        node[i].start = edgeNumber;
        node[i].length = 2;

        // Moving 0 to the left
        Temp[posx][posy] = Temp[posx][posy-1];
        Temp[posx][posy-1] = 0;

        string s1 = "abcd";

        for(int j=0;j<2;j++)
        {
        	for(int k=0;k<2;k++)
        	{
        		s1[j*2+k] = Temp[j][k]+'0';
        	}
        }

        for(int j=0;j<24;j++)
        {
        	if (map[j].s == s1)
        	{
        		edges[edgeNumber++] = map[j].id;
        	}
        }


        // int Temp[2][2];
        for(int j=0;j<2;j++)
        {
        	for(int k=0;k<2;k++)
        	{
        		Temp[j][k] = mat[j][k];
        	}
        }

        // Moving 0 to the bottom
        Temp[posx][posy] = Temp[posx+1][posy];
        Temp[posx+1][posy] = 0;

        // s1 = "abcd";

        for(int j=0;j<2;j++)
        {
        	for(int k=0;k<2;k++)
        	{
        		s1[j*2+k] = Temp[j][k]+'0';
        	}
        }

        for(int j=0;j<24;j++)
        {
        	if (map[j].s == s1)
        	{
        		edges[edgeNumber++] = map[j].id;
        	}
        }
    }

    if (posx == 1 && posy == 0) 
	 {
        int Temp[2][2];
        for(int j=0;j<2;j++)
        {
        	for(int k=0;k<2;k++)
        	{
        		Temp[j][k] = mat[j][k];
        	}
        }

        node[i].start = edgeNumber;
        node[i].length = 2;

        // Moving 0 to the right
        Temp[posx][posy] = Temp[posx][posy+1];
        Temp[posx][posy+1] = 0;

        string s1 = "abcd";

        for(int j=0;j<2;j++)
        {
        	for(int k=0;k<2;k++)
        	{
        		s1[j*2+k] = Temp[j][k]+'0';
        	}
        }

        for(int j=0;j<24;j++)
        {
        	if (map[j].s == s1)
        	{
        		edges[edgeNumber++] = map[j].id;
        	}
        }


        // int Temp[2][2];
        for(int j=0;j<2;j++)
        {
        	for(int k=0;k<2;k++)
        	{
        		Temp[j][k] = mat[j][k];
        	}
        }

        // Moving 0 to the top
        Temp[posx][posy] = Temp[posx-1][posy];
        Temp[posx-1][posy] = 0;

        // s1 = "abcd";

        for(int j=0;j<2;j++)
        {
        	for(int k=0;k<2;k++)
        	{
        		s1[j*2+k] = Temp[j][k]+'0';
        	}
        }

        for(int j=0;j<24;j++)
        {
        	if (map[j].s == s1)
        	{
        		edges[edgeNumber++] = map[j].id;
        	}
        }
    }
    if (posx == 1 && posy == 1) 
	 {
        int Temp[2][2];
        for(int j=0;j<2;j++)
        {
        	for(int k=0;k<2;k++)
        	{
        		Temp[j][k] = mat[j][k];
        	}
        }

        node[i].start = edgeNumber;
        node[i].length = 2;

        // Moving 0 to the left
        Temp[posx][posy] = Temp[posx][posy-1];
        Temp[posx][posy-1] = 0;

        string s1 = "abcd";

        for(int j=0;j<2;j++)
        {
        	for(int k=0;k<2;k++)
        	{
        		s1[j*2+k] = Temp[j][k]+'0';
        	}
        }

        for(int j=0;j<24;j++)
        {
        	if (map[j].s == s1)
        	{
        		edges[edgeNumber++] = map[j].id;
        	}
        }


        // int Temp[2][2];
        for(int j=0;j<2;j++)
        {
        	for(int k=0;k<2;k++)
        	{
        		Temp[j][k] = mat[j][k];
        	}
        }

        // Moving 0 to the top
        Temp[posx][posy] = Temp[posx-1][posy];
        Temp[posx-1][posy] = 0;

        // s1 = "abcd";

        for(int j=0;j<2;j++)
        {
        	for(int k=0;k<2;k++)
        	{
        		s1[j*2+k] = Temp[j][k]+'0';
        	}
        }

        for(int j=0;j<24;j++)
        {
        	if (map[j].s == s1)
        	{
        		edges[edgeNumber++] = map[j].id;
        	}
        }
    }
} 

// The BFS frontier corresponds to all the nodes being processed at the current level.

int main()
{

	// HashMap map[24];
	// map[0].id = 0;
	// map[0].s = '0123';


	// for(int )
	permute("2103", 0, 3);

	for(int i=0;i<24;i++)
	{
		cout<<map[i].id<<" "<<map[i].s<<endl;
	}

	// getneighbour("0123", 0);

	
	
	//int edgesSize = 2 * NUM_NODES;

	for(int i=0;i<24;i++)
	{
		string s = map[i].s;
		int id = map[i].id;
		getneighbour(s, i);
	}

	// node[0].start = 0;
	// node[0].length = 2;

	// node[1].start = 2;
	// node[1].length = 2;

	// node[2].start = 4;
	// node[2].length = 2;

	// node[3].start = 6;
	// node[3].length = 2;

	// node[4].start = 5;
	// node[4].length = 0;

	// edges[0] = 1;
	// edges[1] = 2;	
	// edges[2] = 0;
	// edges[3] = 3;
	// edges[4] = 0;
	// edges[5] = 3;
	// edges[6] = 1;
	// edges[7] = 2;

	bool frontier[NUM_NODES] = { false };
	bool visited[NUM_NODES] = { false };
	int cost[NUM_NODES] = { 0 };

	int source = 0;
	frontier[source] = true;

	Node* Va;
	hipMalloc((void**)&Va, sizeof(Node)*NUM_NODES);
	hipMemcpy(Va, node, sizeof(Node)*NUM_NODES, hipMemcpyHostToDevice);

	int* Ea;
	hipMalloc((void**)&Ea, sizeof(Node)*NUM_NODES);
	hipMemcpy(Ea, edges, sizeof(Node)*NUM_NODES, hipMemcpyHostToDevice);

	bool* Fa;
	hipMalloc((void**)&Fa, sizeof(bool)*NUM_NODES);
	hipMemcpy(Fa, frontier, sizeof(bool)*NUM_NODES, hipMemcpyHostToDevice);

	bool* Xa;
	hipMalloc((void**)&Xa, sizeof(bool)*NUM_NODES);
	hipMemcpy(Xa, visited, sizeof(bool)*NUM_NODES, hipMemcpyHostToDevice);

	int* Ca;
	hipMalloc((void**)&Ca, sizeof(int)*NUM_NODES);
	hipMemcpy(Ca, cost, sizeof(int)*NUM_NODES, hipMemcpyHostToDevice);

	

	int num_blks = 1;
	int threads = 32;



	bool done;
	bool* d_done;
	hipMalloc((void**)&d_done, sizeof(bool));
	printf("\n\n");
	int count = 0;

	printf("Order: \n\n");
	do {
		count++;
		done = true;
		hipMemcpy(d_done, &done, sizeof(bool), hipMemcpyHostToDevice);
		CUDA_BFS_KERNEL <<<num_blks, threads >>>(Va, Ea, Fa, Xa, Ca,d_done);
		hipMemcpy(&done, d_done , sizeof(bool), hipMemcpyDeviceToHost);

	} while (!done);




	hipMemcpy(cost, Ca, sizeof(int)*NUM_NODES, hipMemcpyDeviceToHost);
	
	printf("Number of times the kernel is called : %d \n", count);


	printf("\nCost: ");
	for (int i = 0; i<NUM_NODES; i++)
		printf( "%d    ", cost[i]);
	printf("\n");
	// _getch();
	
}
